#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "typedef.h"
#include "enc.h"
#include "dec.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#define PI 3.14


static const char* pass = "pass";
static const char* fail = "fail";

#define AMR_MAGIC_NUMBER "#!AMR-WB\n"

static const char* name[] = { "Word8", "UWord8", "Word16", "Word32", "Float32",
"Float64" };


__device__ static const unsigned long size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
static const unsigned long h_size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
__global__ void amrinit(enc_interface_State *enstate,dec_interface_State *destate){
       	int dtx = 0;
	Encoder_Interface_init(enstate, dtx);
        Decoder_Interface_init(destate);
       
}
__global__ void amrenc(enc_interface_State *enstate,dec_interface_State *destate,short *speech,unsigned char *serial_data,unsigned char *amrdata) {
	int i=0, j;
        int req_mode = 7;
	int byte_counter = Encoder_Interface_Encode(enstate, Mode(req_mode), speech, serial_data, 0);
        for(i=0;i<32;i++)
        amrdata[i]=serial_data[i];
	//int dec_mode = (serial_data[0] >> 3) & 0x000F;
	//int read_size = block_size[dec_mode];
	Decoder_Interface_Decode(destate, serial_data, speech, 0);
        

}
int main() {
	int i, j;
	clock_t start, finish;

	double  duration;
	for (i = 0; i < 6; i++) {
		const char* result = (h_size[i][0] == h_size[i][1] ? pass : fail);
		printf("%s size: %lu, %s\n", name[i], h_size[i][0], result);
		if (result == fail) {
			exit(1);
		}

	}
        enc_interface_State *d_enstate;
        dec_interface_State *d_destate;	
        hipMalloc(&d_enstate, sizeof(enc_interface_State));
        hipMalloc(&d_destate, sizeof(dec_interface_State));
        amrinit<<<1,1>>>(d_enstate,d_destate);      
        short *speech=new short[160];
        short *d_speech;
	for (j = 0; j < 160; j++) {
        speech[j] = 10460*sin(PI*j/20);
        }
        for(i=0;i<160;i++)
        printf("%d ",speech[i]);
        printf("\n\n");
        hipMalloc(&d_speech, 160*sizeof(short));
        hipMemcpy(d_speech, speech, 160*sizeof(short), hipMemcpyHostToDevice);
        unsigned char *serial_data=new unsigned char[32];
        unsigned char *amrdata=new unsigned char[32];
        unsigned char *d_serial_data;
        unsigned char *d_amrdata;
        hipMalloc(&d_serial_data, 32*sizeof(unsigned char));
        hipMalloc(&d_amrdata, 32*sizeof(unsigned char));
       	start = clock();
	amrenc <<<100, 100 >>>(d_enstate,d_destate,d_speech,d_serial_data,d_amrdata);
        hipMemcpy(speech, d_speech, 160*sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(amrdata, d_amrdata, 32*sizeof(unsigned char), hipMemcpyDeviceToHost);
	finish = clock();
	duration = (double)(finish - start)*1000000 / CLOCKS_PER_SEC;
	printf("%1.1f us\n", duration);
        for(i=0;i<32;i++)
        printf("%d,",amrdata[i]);
        printf("\n");
        printf("\n");
        for(i=0;i<160;i++)
        printf("%d,",speech[i]);
        printf("\n");
	//fclose(pcm_orig);
	//fclose(amrnb);
	//fclose(pcm_back);
	return 0;
}
