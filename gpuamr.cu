#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "typedef.h"
#include "enc.h"
#include "dec.h"
#include "iostream"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstring>
#include <alsa/asoundlib.h>
using namespace std;

#define PI 3.1415
#define CIRTIME 1
#define CIREE 1
#define PERTH 100

static const char* pass = "pass";
static const char* fail = "fail";

#define AMR_MAGIC_NUMBER "#!AMR-WB\n"

static const char* name[] = { "Word8", "UWord8", "Word16", "Word32", "Float32",
"Float64" };


__device__ static const unsigned long size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
static const unsigned long h_size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
int departwavinit(int n,FILE **p)//拆分wav头到达正常读取位置并返回文件地址矩阵的地址**p
{
}
int departwavget(int n,FILE **p,short *speech)
{
}
int makewavinit(int n,FILE **p)//合成amr头到达正常读取位置并返回文件地址矩阵的地址**p
{
}
int makewavpush(int n,FILE **p,short *amrdata)
{
}
int makeamrinit(int n,FILE **p)//合成wav头到达正常读取位置并返回文件地址矩阵的地址**p
{
}
int makeamrpush(int n,FILE **p,short *speech)
{
}
int exmicinit()//麦克风输入初始化
{
}
int exmicget()//麦克风输入
{
}
//播放器初始化
int bcastinit(snd_pcm_uframes_t *frames,snd_pcm_t *playback_handle,snd_pcm_hw_params_t *hw_params)
{
    int dir=0;
    int ret;   
    unsigned int val; 
//1. 打开PCM，最后一个参数为0意味着标准配置 
ret = snd_pcm_open(&playback_handle, "default", SND_PCM_STREAM_PLAYBACK, 0);  
    if (ret < 0) {  
        perror("snd_pcm_open");  
        exit(1);  
    }  
//2. 分配snd_pcm_hw_params_t结构体
ret = snd_pcm_hw_params_malloc(&hw_params);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_malloc");  
        exit(1);  
    }  
//3. 初始化hw_params  
    ret = snd_pcm_hw_params_any(playback_handle, hw_params);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_any");  
        exit(1);  
    }  
    //4. 初始化访问权限  
    ret = snd_pcm_hw_params_set_access(playback_handle, hw_params, SND_PCM_ACCESS_RW_INTERLEAVED);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_set_access");  
        exit(1);  
    }  
    //5. 初始化采样格式SND_PCM_FORMAT_U8,16位  
    ret = snd_pcm_hw_params_set_format(playback_handle, hw_params,SND_PCM_FORMAT_S16_LE);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_set_format");  
        exit(1);  
    }  
    //6. 设置采样率，如果硬件不支持我们设置的采样率，将使用最接近的  
    val = 8000;  
    ret = snd_pcm_hw_params_set_rate_near(playback_handle, hw_params, &val, &dir);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_set_rate_near");  
        exit(1);  
    }  
    //7. 设置通道数量  
    ret = snd_pcm_hw_params_set_channels(playback_handle, hw_params, 1);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params_set_channels");  
        exit(1);  
    }  
    /* Set period size to 32 frames. */  
    *frames = 160;  
    ret = snd_pcm_hw_params_set_buffer_size_near(playback_handle, hw_params, frames);  
  if (ret < 0)   
    {  
        printf("Unable to set period size %li : %s\n", frames,  snd_strerror(ret));  
    }  
    ret = snd_pcm_hw_params_set_period_size_near(playback_handle, hw_params, frames, 0);  
    if (ret < 0)   
    {  
        printf("Unable to set period size %li : %s\n", frames,  snd_strerror(ret));  
    }  
                                    
    //8. 设置hw_params  
    ret = snd_pcm_hw_params(playback_handle, hw_params);  
    if (ret < 0) {  
        perror("snd_pcm_hw_params");  
        exit(1);  
    }  
      
     /* Use a buffer large enough to hold one period */  
    snd_pcm_hw_params_get_period_size(hw_params, frames, &dir); 
    printf("frames size is %d\n",*frames);

}
__global__ void amrinit(enc_interface_State *enstate,dec_interface_State *destate){
        int nx;
       	int dtx = 0;
        nx=threadIdx.x+blockIdx.x*blockDim.x;
        enc_interface_State c_enstate=enstate[nx];
        dec_interface_State c_destate=destate[nx];
	Encoder_Interface_init(&c_enstate, dtx);
        Decoder_Interface_init(&c_destate);
        destate[nx]=c_destate;
        enstate[nx]=c_enstate;
}
__global__ void amrenc(enc_interface_State *enstate,dec_interface_State *destate,short *speech,short *speechout,unsigned char *serial_data,unsigned char *amrdata) {
	int i=0,j;
        int req_mode = 7;
        int nx;
        nx=threadIdx.x+blockIdx.x*blockDim.x;

	int byte_counter = Encoder_Interface_Encode(&enstate[nx], Mode(req_mode), &speech[160*nx], &amrdata[nx*32], 0);

        for(i=0;i<32;i++)
        serial_data[nx*32+i]=amrdata[nx*32+i];

	Decoder_Interface_Decode(&destate[nx], &amrdata[nx*32], &speechout[160*nx], 0);
        //printf("the nx is %d\n",nx);
}
__global__ void amrprint(short * speech)
{
  int i,j;
  for(i=0;i<CIRTIME*PERTH;i++){
  for(j=0;j<160;j++){
  printf("%d ",speech[i*160+j]);
}
  printf("\n");
}
}
/*int main() {
        //FILE* pcm_back = fopen("cclnb.md", "wb");
	int i, j;
	clock_t start, finish;

	double  duration;
	for (i = 0; i < 6; i++) {
		const char* result = (h_size[i][0] == h_size[i][1] ? pass : fail);
		printf("%s size: %lu, %s\n", name[i], h_size[i][0], result);
		if (result == fail) {
			exit(1);
		}

	}
        enc_interface_State *d_enstate;
        dec_interface_State *d_destate;	
        hipMalloc(&d_enstate, CIRTIME*sizeof(enc_interface_State));
        hipMalloc(&d_destate, CIRTIME*sizeof(dec_interface_State));
        dim3 block(CIRTIME/CIREE,1);
        dim3 grid(CIREE,1);
        amrinit<<<grid,block>>>(d_enstate,d_destate);
      
        short speech [CIRTIME*PERTH*160];
        short *d_speech;
        for (i = 0; i < CIRTIME*PERTH; i++)
	for (j = 0; j < 160; j++) 
        speech[i*160+j] = 10460*sin(PI*(i*160+j)/48);

        hipMalloc(&d_speech, 160*CIRTIME*PERTH*sizeof(short));
        hipMemcpy(d_speech,speech, 160*CIRTIME*PERTH*sizeof(short), hipMemcpyHostToDevice);

        //amrprint<<<1,1>>>(d_speech);
        unsigned char serial_data[CIRTIME*PERTH*32];
        unsigned char amrdata[CIRTIME*PERTH*32];
        unsigned char *d_serial_data;
        unsigned char *d_amrdata;
        hipMalloc(&d_serial_data, 32*CIRTIME*PERTH*sizeof(unsigned char));
        hipMalloc(&d_amrdata, 32*CIRTIME*PERTH*sizeof(unsigned char));
        //amrenc <<<1, 1 >>>(d_enstate,d_destate,d_speech,d_serial_data,d_amrdata);

       	start = clock();
        hipDeviceSynchronize();
	amrenc <<<grid, block >>>(d_enstate,d_destate,d_speech,d_serial_data,d_amrdata);
        hipMemcpy(speech, d_speech, 160*CIRTIME*PERTH*sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(serial_data, d_serial_data, 32*CIRTIME*PERTH*sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
	finish = clock();
        for (i = 0; i < CIRTIME*PERTH; i++) {
        for(j=0;j<32;j++)
        printf("%X,",serial_data[i*32+j]);
        printf("\n");}
        printf("\n");
        for(i=0;i<CIRTIME*PERTH;i++){
        for(j=0;j<160;j++){
        printf("%d ",speech[i*160+j]);
        }
        printf("\n");
        }
	duration = (double)(finish - start)*1000000 / CLOCKS_PER_SEC;
	printf("%1.1f us\n", duration);
        //fwrite(speech, sizeof(short int), 160*CIRTIME, pcm_back);
	//fclose(pcm_orig);
	//fclose(amrnb);
	//fclose(pcm_back);
	return 0;
}*/
int printamr(unsigned char *amr,int n)
{
        int i,j;
        for(i=0;i<n;i++){
        for(j=0;j<32;j++)
        printf("%X,",amr[i*32+j]);
        printf("\n");}
        printf("\n");
}
int printpcm(short *pcm,int n)
{
        int i,j;
        for(i=0;i<n;i++){
        for(j=0;j<160;j++)
        printf("%d,",pcm[i*160+j]);
        printf("\n");}
        printf("\n");
}
int main()
{
        int i=0,j=0,flagc=0,flagd=0;
	clock_t start, finish;
	double  duration;
	for (i = 0; i < 6; i++) {
		const char* result = (h_size[i][0] == h_size[i][1] ? pass : fail);
		printf("%s size: %lu, %s\n", name[i], h_size[i][0], result);
		if (result == fail) {
			exit(1);
		}

	}
        //数据测试
        char *buf;
        int ret;
        int size=320;
        unsigned char amr[(PERTH+3)*32];
        short speechdata [(PERTH+3)*160];
        short speechdataout [(PERTH+3)*160];
        for (i = 0; i < 8000; i++) 
        speechdata[i] = 10460*sin(PI*i/49);
        //初始化播放器
        snd_pcm_uframes_t *frames=new snd_pcm_uframes_t;  
        snd_pcm_t *playback_handle;//PCM设备句柄pcm.h  
        snd_pcm_hw_params_t *hw_params;//硬件信息和PCM流配置 
        bcastinit(frames,playback_handle,hw_params);
        //在GPU上初始化enstate与destate
        dim3 block(CIRTIME/CIREE,1);
        dim3 grid(CIREE,1);
        enc_interface_State *d_enstate;
        dec_interface_State *d_destate;	
        hipMalloc(&d_enstate, CIRTIME*sizeof(enc_interface_State));
        hipMalloc(&d_destate, CIRTIME*sizeof(dec_interface_State));
        amrinit<<<grid,block>>>(d_enstate,d_destate);
        //双缓冲结构
        short speech_1 [CIRTIME*160];
        short speech_2 [CIRTIME*160];
        short speechout_1 [CIRTIME*160];
        short speechout_2 [CIRTIME*160];
        unsigned char serial_data_1[CIRTIME*32];
        unsigned char amrdata_1[CIRTIME*32];
        unsigned char serial_data_2[CIRTIME*32];
        unsigned char amrdata_2[CIRTIME*32];
        //显存开辟与内存指针
        short *d_speech1;
        short *d_speechout1;
        unsigned char *d_amrdata1;
        unsigned char *d_serial_data1;
        hipMalloc(&d_serial_data1, 32*CIRTIME*sizeof(unsigned char));
        hipMalloc(&d_amrdata1, 32*CIRTIME*sizeof(unsigned char));
        hipMalloc(&d_speech1, 160*CIRTIME*sizeof(short));
        hipMalloc(&d_speechout1, 160*CIRTIME*sizeof(short));
        short *d_speech2;
        short *d_speechout2;
        unsigned char *d_serial_data2;
        unsigned char *d_amrdata2;
        hipMalloc(&d_serial_data2, 32*CIRTIME*sizeof(unsigned char));
        hipMalloc(&d_amrdata2, 32*CIRTIME*sizeof(unsigned char));
        hipMalloc(&d_speech2, 160*CIRTIME*sizeof(short));
        hipMalloc(&d_speechout2, 160*CIRTIME*sizeof(short));
        //初始化文件
        FILE* wav_input[CIRTIME];
        FILE* amr_output[CIRTIME];
        FILE* wav_output[CIRTIME];
        //departwavinit(CIRTIME,wav_input);
        //makewavinit(CIRTIME,wav_output);
        //makeamrinit(CIRTIME,amr_output);
        //外循环控制由CPU负责完成，计算由GPU负责完成
      buf=(char*)malloc(size);
      for(i=0;i<PERTH+1;i++)
      {
        if(i%2==0){
        //departwavget(CIRTIME,wav_input,speech_1);
        memcpy(speech_1,&speechdata[i*160],160*sizeof(short));
        hipMemcpyAsync(d_speech1, speech_1, 160*CIRTIME*sizeof(short), hipMemcpyHostToDevice,0);
        if(flagc==1){
        amrenc <<<grid, block >>>(d_enstate,d_destate,d_speech2,d_speechout2,d_serial_data2,d_amrdata2);//GPU
        hipMemcpyAsync(speechout_1, d_speechout1, 160*CIRTIME*sizeof(short), hipMemcpyDeviceToHost,0);
        hipMemcpyAsync(serial_data_1,d_serial_data1, 32*CIRTIME*sizeof(unsigned char), hipMemcpyDeviceToHost,0);
         memcpy(&amr[i*32],serial_data_1,32*sizeof(unsigned char));
        memcpy(&speechdataout[i*160],speechout_1,160*sizeof(short));
                memcpy(buf,speechout_1,size);
        while(ret=snd_pcm_writei(playback_handle,buf, *frames)<0)
             {
             if(ret == -EPIPE)
                {
                snd_pcm_prepare(playback_handle);
                }
             else if(ret < 0)
                {
                snd_strerror(ret);
                }
             }
        //makewavpush(CIRTIME,wav_output,speech_2);
        //makeamrpush(CIRTIME,amr_output,serial_data_2);
        flagd=1;
           }
        }
        else{
        //departwavget(CIRTIME,wav_input,speech_2);
        memcpy(speech_2,&speechdata[i*160],160*sizeof(short));
        hipMemcpyAsync(d_speech2, speech_2, 160*CIRTIME*sizeof(short), hipMemcpyHostToDevice,0);
        amrenc <<<grid, block >>>(d_enstate,d_destate,d_speech1,d_speechout1,d_serial_data1,d_amrdata1);//GPU
        if(flagd==1){
        hipMemcpyAsync(speechout_2, d_speechout2, 160*CIRTIME*sizeof(short), hipMemcpyDeviceToHost,0);
        hipMemcpyAsync(serial_data_2, d_serial_data2, 32*CIRTIME*sizeof(unsigned char),hipMemcpyDeviceToHost,0);
        memcpy(&amr[i*32],serial_data_2,32*sizeof(unsigned char));
        memcpy(&speechdataout[i*160],speechout_2,size);
        memcpy(buf,speechout_2,size);
        while(ret=snd_pcm_writei(playback_handle, buf, *frames)<0)
             {
             if(ret == -EPIPE)
                {
                snd_pcm_prepare(playback_handle);
                }
             else if(ret < 0)
                {
                snd_strerror(ret);
                }
             }
        }
        //makewavpush(CIRTIME,wav_output,speech_1);
        //makeamrpush(CIRTIME,amr_output,serial_data_1);
        //
        flagc=1;
        }
      }
        //printamr(amr,48);
        //printpcm(speechdataout,20);
        snd_pcm_close(playback_handle);
        hipFree(d_speech1);
        hipFree(d_speech2);
        hipFree(d_serial_data1);
        hipFree(d_serial_data2);
        hipFree(d_amrdata1);
        hipFree(d_amrdata2);
}

