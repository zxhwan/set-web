#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "typedef.h"
#include "enc.h"
#include "dec.h"
#include "iostream"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstring>
using namespace std;

#define PI 3.1415
#define CIRTIME 1
#define CIREE 1
#define PERTH 100

__device__ static const unsigned long size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
static const unsigned long h_size[][2] = {
	{ sizeof(Word8), 1 },
	{ sizeof(UWord8), 1 },
	{ sizeof(Word16), 2 },
	{ sizeof(Word32), 4 },
	{ sizeof(Float32), 4 },
	{ sizeof(Float64), 8 }
};
/* 函数作用：编码器初始化函数
*  enstate：编码器的继承码本
*/
__global__ void amr_encode_init(enc_interface_State *enstate){
        int nx;
       	int dtx = 0;
        nx=threadIdx.x+blockIdx.x*blockDim.x;
        enc_interface_State c_enstate=enstate[nx];
	Encoder_Interface_init(&c_enstate, dtx);
        enstate[nx]=c_enstate;
}
/* 函数作用：解码器初始化函数
*  destate：解码器的继承码本
*/
__global__ void amr_decode_init(dec_interface_State *destate){
        int nx;
        nx=threadIdx.x+blockIdx.x*blockDim.x;
        dec_interface_State c_destate=destate[nx];
        Decoder_Interface_init(&c_destate);
        destate[nx]=c_destate;
}
/* 函数作用：编码的GPU函数，编码一个并行的内容
*  enstate：编码器码本
*  speech:  PCM码
*  amr_data： AMR码
*/
__global__ void amrenc(enc_interface_State *enstate,short *speech,unsigned char *amr_data) {
        int req_mode = 7;
        int nx;
        nx=threadIdx.x+blockIdx.x*blockDim.x;
	int byte_counter = Encoder_Interface_Encode(&enstate[nx], Mode(req_mode), &speech[160*nx], &amr_data[nx*32], 0);
}
/* 函数作用：解码的GPU函数，编码一个并行的内容
*  enstate：解码器码本
*  speech:  PCM码
*  amr_data： AMR码
*/
__global__ void amrdec(dec_interface_State *destate,unsigned char *amrdata,short *speech_out) {
	int i=0,j;
        int req_mode = 7;
        int nx;
        nx=threadIdx.x+blockIdx.x*blockDim.x;
        Decoder_Interface_Decode(&destate[nx], &amrdata[nx*32], &speech_out[160*nx], 0);
}
__device__ void amrprint(short * speech)
{
  int i,j;
  for(i=0;i<CIRTIME*PERTH;i++){
  for(j=0;j<160;j++){
  printf("%d ",speech[i*160+j]);
   }
  printf("\n");
 }
}
/* speech大小等于（serial+4）*parallel*160个short内容，保证最后4*parallel*160个short为空 */
unsigned char * encode_interface(short *speech, int serial, int parallel){
      unsigned char *amrdata;
      int i,m,n,num;
      int flagc=0,flagd=0;
      amrdata=(unsigned char*)malloc(32*(serial+4)*parallel*sizeof(unsigned char));
      amrdata[1000]=10;
      printf("parallel:%d\n",parallel);
      if((parallel%16!=0)&&(parallel>16)) {
      n=16;
      m=(parallel+(parallel%16))/n;
      }
      else if(parallel<16){
      n=parallel;
      m=1;
      }
      else{
      n=16;
      m=parallel/n;
      }
      printf("block:%d,grid:%d,size of amr data:%d\n",m,n,amrdata[1000]);
      dim3 grid(m,1);
      dim3 block(n,1);
      num=m*n;
      short *speech_1;
      short *speech_2;
      unsigned char *amrdata1;
      unsigned char *amrdata2;
      amrdata1=(unsigned char*)malloc(32*num*sizeof(unsigned char));
      speech_1=(short*)malloc(160*num*sizeof(short));
      amrdata2=(unsigned char*)malloc(32*num*sizeof(unsigned char));
      speech_2=(short*)malloc(160*num*sizeof(short));

      short *d_speech1;
      short *d_speech2;
      unsigned char *d_amrdata1;
      unsigned char *d_amrdata2;
      hipMalloc(&d_amrdata1, 32*num*sizeof(unsigned char));
      hipMalloc(&d_speech1, 160*num*sizeof(short));
      hipMalloc(&d_amrdata2, 32*num*sizeof(unsigned char));
      hipMalloc(&d_speech2, 160*num*sizeof(short));

      enc_interface_State *d_enstate;
      hipMalloc(&d_enstate, num*sizeof(enc_interface_State));
      amr_encode_init<<<grid,block>>>(d_enstate);

      for(i=0;i<serial+2;i++)
        {
        if(i%2==0){
        memcpy(speech_1,&speech[i*parallel*160],160*num*sizeof(short));
        hipMemcpyAsync(d_speech1, speech_1, 160*num*sizeof(short), hipMemcpyHostToDevice,0);
        if(flagc==1){
        amrenc <<<grid, block >>>(d_enstate,d_speech2,d_amrdata2);//GPU
        hipMemcpyAsync(amrdata1,d_amrdata1, 32*num*sizeof(unsigned char), hipMemcpyDeviceToHost,0);
        memcpy(&amrdata[i*parallel*32],amrdata1,32*num*sizeof(short));
        flagd=1;
           }
        }
        else{
        flagc=1;
        memcpy(speech_2,&speech[i*parallel*160],160*num*sizeof(short));
        hipMemcpyAsync(d_speech2, speech_2, 160*num*sizeof(short), hipMemcpyHostToDevice,0);
        amrenc <<<grid, block >>>(d_enstate,d_speech1,d_amrdata1);//GPU
        if(flagd==1){
        hipMemcpyAsync(amrdata2, d_amrdata2, 32*num*sizeof(unsigned char), hipMemcpyDeviceToHost,0);
        memcpy(&amrdata[i*parallel*32],amrdata2,32*num*sizeof(short));
        }
      }
    }
      unsigned char *amrdataout;
      amrdataout=(unsigned char*)malloc(32*(serial+4)*parallel*sizeof(unsigned char));
      memcpy(amrdataout,&amrdata[2*parallel*32],32*parallel*serial*sizeof(unsigned char));
      hipFree(d_speech1);
      hipFree(d_speech2);
      hipFree(d_amrdata1);
      hipFree(d_amrdata2);
      return amrdataout;
}
/* amrdata大小等于（serial+4）*parallel*32个char内容，保证最后4*parallel*160个char为空 */
short * decode_interface(unsigned char *amrdata, int serial, int parallel)
{
      short *speech;
      int flagc,flagd;
      int i,m,n,num;
      speech=(short*)malloc(160*(serial+4)*parallel*sizeof(short));

      if((parallel%16!=0)&&(parallel>16)) {
      n=16;
      m=(parallel+(parallel%16))/n;
      }
      else if(parallel<16){
      n=parallel;
      m=1;
      }
      else{
      n=16;
      m=parallel/n;
      }
      dim3 block(m,1);
      dim3 grid(n,1);
      num=m*n;

      short *speech_1;
      short *speech_2;
      unsigned char *amrdata1;
      unsigned char *amrdata2;
      amrdata1=(unsigned char*)malloc(32*num*sizeof(unsigned char));
      speech_1=(short*)malloc(160*num*sizeof(short));
      amrdata2=(unsigned char*)malloc(32*num*sizeof(unsigned char));
      speech_2=(short*)malloc(160*num*sizeof(short));

      short *d_speech1;
      short *d_speech2;
      unsigned char *d_amrdata1;
      unsigned char *d_amrdata2;
      hipMalloc(&d_amrdata1, 32*num*sizeof(unsigned char));
      hipMalloc(&d_speech1, 160*num*sizeof(short));
      hipMalloc(&d_amrdata2, 32*num*sizeof(unsigned char));
      hipMalloc(&d_speech2, 160*num*sizeof(short));

      dec_interface_State *d_destate;
      hipMalloc(&d_destate, CIRTIME*sizeof(dec_interface_State));
      amr_decode_init<<<grid, block>>>(d_destate);

      for(i=0;i<serial+2;i++)
        {
        if(i%2==0){
        memcpy(amrdata1,&amrdata[i*parallel*32],32*num*sizeof(unsigned char));
        hipMemcpyAsync(d_amrdata1, amrdata1, 32*num*sizeof(unsigned char), hipMemcpyHostToDevice,0);
        if(flagc==1){
        amrdec <<<grid, block >>>(d_destate,d_amrdata2,d_speech2);//GPU
        hipMemcpyAsync(speech_1,d_speech1, 160*num*sizeof(short), hipMemcpyDeviceToHost,0);
        memcpy(&speech[i*parallel*160],speech_1,160*num*sizeof(short));
        flagd=1;
           }
        }
        else{
        flagc=1;
        memcpy(amrdata2,&speech[i*parallel*32],32*num*sizeof(unsigned char));
        hipMemcpyAsync(d_amrdata2, amrdata2, 32*num*sizeof(unsigned char), hipMemcpyHostToDevice,0);
        amrdec <<<grid, block >>>(d_destate,d_amrdata1,d_speech1);//GPU
        if(flagd==1){
        hipMemcpyAsync(speech_1, d_speech1, 160*num*sizeof(short), hipMemcpyDeviceToHost,0);
        memcpy(&speech[i*parallel*160],speech_1,160*num*sizeof(short));
        }
      }
    }
      short *speechout;
      speechout=(short*)malloc(160*(serial+4)*parallel*sizeof(short));
      memcpy(speechout,&speech[2*parallel*160],160*parallel*serial*sizeof(short));
      for(i = 0; i < 10*160; i++)
      printf("%d ",speechout[i]);
      hipFree(d_speech1);
      hipFree(d_speech2);
      hipFree(d_amrdata1);
      hipFree(d_amrdata2);
      return speechout;
}
/**/
__global__ void transpose_PCM(short *odata, short *idata, int width, int height)  
{  
    int tran_x, tran_y, nx;
    nx = threadIdx.x+blockIdx.x*blockDim.x;
    tran_x = nx % width; 
    tran_y = nx / width;
    hipMemcpy(&odata[(height*tran_x+tan_y)*160], &idata[nx*160],160*sizeof(short), hipMemcpyDeviceToDevice);
}  
__global__ void transpose_AMR(unsigned char *odata, unsigned char *idata, int width, int height)  
{  
    int tran_x, tran_y, nx;
    nx = threadIdx.x+blockIdx.x*blockDim.x;
    tran_x = nx % width; 
    tran_y = nx / width;
    hipMemcpy(&odata[(height*tran_x+tan_y)*32], &idata[nx*32],32*sizeof(char), hipMemcpyDeviceToDevice);
}  
int pcmdata_t(short *odata,short *idata, int serial, int parallel, int ds)
{
dim3 block(serial,1);
dim3 grid(parallel,1);
short *h_odata,*h_idata;
hipMalloc(&h_odata,sizeof(idata));
hipMalloc(&h_idata,sizeof(idata));
if(ds==0){
hipMemcpy(h_idata, idata, sizeof(idata), hipMemcpyHostToDevice);
 transpose_PCM <<<grid, block >>>(h_odata,h_idata,serial,parallel);
hipMemcpy(odata, h_odata, sizeof(idata),  hipMemcpyDeviceToHost);
return 0;
 }
if(ds==1){
hipMemcpy(h_idata, idata, sizeof(idata), hipMemcpyHostToDevice);
 transpose_PCM <<<grid, block >>>(h_odata,h_idata,parallel,serial);
hipMemcpy(odata, h_odata, sizeof(idata),  hipMemcpyDeviceToHost);
return 0;
}
else return 1;
}
int amrdata_t(unsigned char *odata,unsigned char *idata, int serial, int parallel, int ds)
{
dim3 block(serial,1);
dim3 grid(parallel,1);
unsigned char  *h_odata,*h_idata;
hipMalloc(&h_odata,sizeof(idata));
hipMalloc(&h_idata,sizeof(idata));
if(ds==0){
hipMemcpy(h_idata, idata, sizeof(idata), hipMemcpyHostToDevice);
 transpose_PCM <<<grid, block >>>(h_odata,h_idata,serial,parallel);
hipMemcpy(odata, h_odata, sizeof(idata),  hipMemcpyDeviceToHost);
return 0;
 }
if(ds==1){
hipMemcpy(h_idata, idata, sizeof(idata), hipMemcpyHostToDevice);
 transpose_PCM <<<grid, block >>>(h_odata,h_idata,parallel,serial);
hipMemcpy(odata, h_odata, sizeof(idata),  hipMemcpyDeviceToHost);
return 0;
}
else return 1;
}
